
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
 
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}
 
int main(void)
{
 int N = 1<<29; // 512M
 float *x, *y;
 
 // Allocate Unified Memory – accessible from CPU or GPU
 hipMallocManaged(&x, N*sizeof(float));
 hipMallocManaged(&y, N*sizeof(float));
 
 // initialize x and y arrays on the host
 for (int i = 0; i < N; i++) {
   x[i] = 1.0f;
   y[i] = 2.0f;
 }
 
 int blockSize = 256;
int numBlocks = (N + blockSize - 1) / blockSize;
printf("numBlocks = %d\n", numBlocks);
add<<<numBlocks, blockSize>>>(N, x, y);
 
 // Wait for GPU to finish before accessing on host
 hipDeviceSynchronize();
 
 // Check for errors (all values should be 3.0f)
 float maxError = 0.0f;
 for (int i = 0; i < N; i++) {
   maxError = fmax(maxError, fabs(y[i]-3.0f));
 }
 std::cout << "Max error: " << maxError << std::endl;
 
 // Free memory
 hipFree(x);
 hipFree(y);
  return 0;
}