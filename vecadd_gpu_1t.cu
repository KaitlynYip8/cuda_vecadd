#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <chrono>
 
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *sum, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    sum[i] = x[i] + y[i];
}
 
int main(void)
{
 int N = 1<<29; // 512M elements
 
 // Allocate Unified Memory -- accessible from CPU or GPU
float *x, *y, *sum;
hipMallocManaged(&x, N*sizeof(float));
hipMallocManaged(&y, N*sizeof(float));

 
// Run kernel on 1M elements on the GPU
add<<<1, 1>>>(N, x, y);
 
// Wait for GPU to finish before accessing on host
hipDeviceSynchronize();
 
 // Check for errors (all values should be 3.0f)
 float maxError = 0.0f;
 for (int i = 0; i < N; i++)
   maxError = fmax(maxError, fabs(y[i]-3.0f));
 std::cout << "Max error: " << maxError << std::endl;
 
// Free memory
hipFree(x);
hipFree(y);
 
 return 0;
}
